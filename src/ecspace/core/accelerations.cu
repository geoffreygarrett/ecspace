#include "hip/hip_runtime.h"
//#include "timer.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_SIZE 256

#include "components.h"

typedef struct {
    float4 *pos, *vel;
} BodySystem;

void randomizeBodies(float *data, int n) {
    for (int i = 0; i < n; i++) {
        data[i] = 2.0f * (rand() / (float) RAND_MAX) - 1.0f;
    }
}


//__global__ void calculate_derivatives(

__global__ void point_mass_force(float4 *p, float4 *v, float dt, int n) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) {
        float Fx = 0.0f;
        float Fy = 0.0f;
        float Fz = 0.0f;

        for (int j = 0; j < n; j++) {
            float dx = p[j].x - p[i].x;
            float dy = p[j].y - p[i].y;
            float dz = p[j].z - p[i].z;
            float distSqr = dx * dx + dy * dy + dz * dz + SOFTENING;
            float invDist = rsqrtf(distSqr);
            float invDist3 = invDist * invDist * invDist;

            Fx += dx * invDist3;
            Fy += dy * invDist3;
            Fz += dz * invDist3;
        }

        v[i].x += dt * Fx;
        v[i].y += dt * Fy;
        v[i].z += dt * Fz;
    }
}

__global__ void bodyForce(float4 *p, float4 *v, float dt, int n) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) {
        float Fx = 0.0f;
        float Fy = 0.0f;
        float Fz = 0.0f;

        for (int j = 0; j < n; j++) {
            float dx = p[j].x - p[i].x;
            float dy = p[j].y - p[i].y;
            float dz = p[j].z - p[i].z;
            float distSqr = dx * dx + dy * dy + dz * dz + SOFTENING;
            float invDist = rsqrtf(distSqr);
            float invDist3 = invDist * invDist * invDist;

            Fx += dx * invDist3;
            Fy += dy * invDist3;
            Fz += dz * invDist3;
        }

        v[i].x += dt * Fx;
        v[i].y += dt * Fy;
        v[i].z += dt * Fz;
    }
}
