#include "hip/hip_runtime.h"
/// Introduction
/// - https://developer.nvidia.com/blog/easy-introduction-cuda-c-and-c/
///
#include "helper.cuh"
#include <iostream>
#include <math.h>
#include <stdio.h>


/// \brief CUDA Kernel function to add the elements of two arrays on the GPU
/// \notes
/// 1. __global__ functions are known as `kernels`, and code that runs on the device is known as `device code`.
/// 2. The __global__ keyword tells the compiler that this function will be called from the host, but executed on the device.
__global__
void add(int n, float *x, float *y)
{
    for (int i = 0; i < n; i++)
        y[i] = x[i] + y[i];
}

/// Memory Allocation in CUDA
/// \long_description
/// To compute on the GPU, I need to allocate memory accessible by the GPU.
/// Unified Memory in CUDA makes this easy by providing a single memory space
/// accessible by all GPUs and CPUs in your system. To allocate data in unified
/// memory, call hipMallocManaged(), which returns a pointer that you can access
/// from host (CPU) code or device (GPU) code. To free the data, just pass
/// the pointer to hipFree().
///
/// I just need to replace the calls to new in the code above with calls
/// to hipMallocManaged(), and replace calls to delete [] with calls to hipFree.


int main(void)
{

    int N = 1<<20;
    float *x, *y;

    // Prints helper device info stuff
    device_info();

    // Allocate Unified Memory – accessible from CPU or GPU
    hipMallocManaged(&x, N*sizeof(float));
    hipMallocManaged(&y, N*sizeof(float));

    // initialize x and y arrays on the host
    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    hipError_t errSync  = hipGetLastError();
    hipError_t errAsync = hipDeviceSynchronize();
    if (errSync != hipSuccess)
        printf("[1] Sync kernel error: %s\n", hipGetErrorString(errSync));
    if (errAsync != hipSuccess)
        printf("[1] Async kernel error: %s\n", hipGetErrorString(errAsync));

    // Run kernel on 1M elements on the GPU
    add<<<1, 1>>>(N, x, y);

    /// \error_checking
    /// This code checks for both synchronous and asynchronous errors.
    /// Invalid execution configuration parameters, e.g. too many threads
    /// per thread block, are reflected in the value of errSync returned
    /// by hipGetLastError(). Asynchronous errors that occur on the device
    /// after control is returned to the host, such as out-of-bounds memory
    /// accesses, require a synchronization mechanism such as
    /// hipDeviceSynchronize(), which blocks the host thread until all
    /// previously issued commands have completed. Any asynchronous error
    /// is returned by hipDeviceSynchronize(). We can also check for
    /// asynchronous errors and reset the runtime error state by modifying
    /// the last statement to call hipGetLastError().
    hipError_t errSync2  = hipGetLastError();
    hipError_t errAsync2 = hipDeviceSynchronize();
    if (errSync2 != hipSuccess)
        printf("[2] Sync kernel error: %s\n", hipGetErrorString(errSync2));
    if (errAsync2 != hipSuccess)
        printf("[2] Async kernel error: %s\n", hipGetErrorString(errAsync2));

    /// \note
    /// Device synchronization is expensive, because it causes the entire
    /// device to wait, destroying any potential for concurrency at that
    /// point in your program. So use it with care. Typically, I use
    /// preprocessor macros to insert asynchronous error checking only
    /// in debug builds of my code, and not in release builds.

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    // Check for errors (all values should be 3.0f)
    float maxError = 0.0f;
    for (int i = 0; i < N; i++)
        maxError = fmax(maxError, fabs(y[i]-3.0f));
    std::cout << "Max error: " << maxError << std::endl;

    // Free memory
    hipFree(x);
    hipFree(y);

    return 0;
}